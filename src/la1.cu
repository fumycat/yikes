#include "utils.h"


int main(int argc, char const* argv[])
{
    /*
        1. int m - A matrix rows & y vector len
        2. int n - A matrix columns & x vector len

        3. float alpha
        4. float beta
        5. const char* - A matrix file
        6. const char* - x vector file
        7. const char* - y vector file
        8. const char* - output vector file
        
        example:
        ./e 512 256 A.txt x.txt y.txt out.txt

        A: m*n
        x: n
        y: m
    */
    int m = atoi(argv[1]);
    int n = atoi(argv[2]);

    float alpha = atof(argv[3]);
    float beta = atof(argv[4]);

    std::ifstream f0(argv[5]);
    std::ifstream f1(argv[6]);
    std::ifstream f2(argv[7]);
    std::ofstream f3(argv[8]);

    hipblasHandle_t handle;

    float* a;
    float* x;
    float* y;

    a = (float*)malloc(m * n * sizeof(float));
    x = (float*)malloc(n * sizeof(float));
    y = (float*)malloc(m * sizeof(float));

    // read files
    float t;
    int i, j;
    for (i = 0; i < m; i++) {
        for (j = 0; j < n; j++) {
            f0 >> t;
            a[i * n + j] = t;
        }
    }
    for (i = 0; i < n; i++) {
        f1 >> t;
        x[i] = t;
    }
    for (i = 0; i < m; i++) {
        f2 >> t;
        y[i] = t;
    }

    float* d_a;
    float* d_x;
    float* d_y;

    // alocate memory on device
    HANDLE_ERROR(hipMalloc((void**)&d_a, m * n * sizeof(*a)));
    HANDLE_ERROR(hipMalloc((void**)&d_x, n * sizeof(*x)));
    HANDLE_ERROR(hipMalloc((void**)&d_y, m * sizeof(*y)));

    // create context
    HANDLE_BLERR(hipblasCreate(&handle));
    
    // copy host to device
    HANDLE_BLERR(hipblasSetMatrix(m, n, sizeof(*a), a, m, d_a, m));
    HANDLE_BLERR(hipblasSetVector(n, sizeof(*x), x, 1, d_x, 1));
    HANDLE_BLERR(hipblasSetVector(m, sizeof(*y), y, 1, d_y, 1));

    // gemv
    HANDLE_BLERR(hipblasSgemv(handle, HIPBLAS_OP_N, m, n, &alpha, d_a, m, d_x, 1, &beta, d_y, 1));

    // copy device to host
    HANDLE_BLERR(hipblasGetVector(m, sizeof(*y), d_y, 1, y, 1));

    // write file
    for (i = 0; i < m; i++) {
        #ifdef FLOAT_IO_REPR
        f3 << FLOAT_IO_REPR << y[i];
        #else
        f3 << y[i];
        #endif
        if (i == m - 1) 
            f3 << std::endl;
        else
            f3 << " ";
    }

    hipFree(d_a);
    hipFree(d_x);
    hipFree(d_y);
    hipblasDestroy(handle);

    return 0;
}
