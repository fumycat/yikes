#include <iostream>

using namespace std;

void open_read_input_file(const char* filename)
{
    cout << filename << endl;
}

int main(int argc, const char *argv[])
{
    // ./e out_name input0 input1 ...
    // 0   1        2      3      ...
    if (argc == 1)
    {
        cout << "input files not specified" << endl;
        cout << "example: " << argv[0] << " output.txt input0.txt input1.txt ..." << endl;
        return -1;
    }
    cout << "Output: " << argv[1] << endl;
    for (int i = 2; i < argc; ++i)
    {
        open_read_input(argv[i]);
    }
    return 0;
}