#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

using namespace std;


__global__ void multKernel(float* A, float* B, float* C, int n) {

    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    float tmpSum = 0;

    if (row < n && col < n) {
        for (int i = 0; i < n; i++) {
            tmpSum += A[row * n + i] * B[i * n + col];
        }
    }
    C[row * n + col] = tmpSum;
}


int main(int argc, char const *argv[])
{
    if (argc < 3) {
        cout << "Input file not specified. Please, specify it as a first argument." << endl;
        cout << "example: " << argv[0] << "file0.txt file1.txt" << endl;
        return -1;
    }

    ifstream f0(argv[1]);
    ifstream f1(argv[2]);
    ofstream output("out.txt");
    if (!f0) {
        cout << "f0 err" << endl;
        return -1;
    }
    if (!f1) {
        cout << "f1 err" << endl;
        return -1;
    }

    int n, zn;
    f0 >> n;
    f1 >> zn;
    if (n != zn) {
        cout << "dim error" << endl;
        return -1;
    }

    thrust::host_vector<float> host_a(n * n), host_b(n * n);
    int i = 0;
    float t;
    while (f0 >> t) {
        host_a[i++] = t;
    }
    i = 0;
    while (f1 >> t) {
        host_b[i++] = t;
    }

    thrust::device_vector<float> mat_a(n * n), mat_b(n * n), mat_out(n * n);

    // copy host to device
    mat_a = host_a;
    mat_b = host_b;

    dim3 threadsPerBlock(n, n);
    dim3 blocksPerGrid(1, 1);
    if (n*n > 512){
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        blocksPerGrid.x = ceil(double(n)/double(threadsPerBlock.x));
        blocksPerGrid.y = ceil(double(n)/double(threadsPerBlock.y));
    }
    multKernel<<<blocksPerGrid,threadsPerBlock>>>(thrust::raw_pointer_cast(&mat_a[0]), thrust::raw_pointer_cast(&mat_b[0]), thrust::raw_pointer_cast(&mat_out[0]), n);


    thrust::host_vector<float> host_out(n * n);
    host_out = mat_out;

    output << n << endl;
    for (int k = 0; k < n; ++k)
    {
        for ( int j = 0; j < n; ++j)
        {
            output << host_out[k * n + j] << " ";
        }
        output << endl;
    }

    // auto delete when function returns
    return 0;
}