#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <iostream>
#include <fstream>

// #include "functor.h"

int main(int argc, char const *argv[])
{
    std::ofstream fs("values.txt");

    thrust::device_vector<float> X(1000000);
    thrust::device_vector<float> Y(1000000);

    thrust::sequence(X.begin(), X.end());
    thrust::fill(Y.begin(), Y.end(), 1.5);

    // thrust::sequence(Y.begin(), Y.end());

    // thrust::transform(X.begin(), X.end(), Y.begin(), thrust::negate<int>());
    // float A = 1.0;

    thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), thrust::multiplies<float>());

    thrust::copy(Y.begin(), Y.end(), std::ostream_iterator<float>(fs, " "));

}
