#include "utils.h"


int main(int argc, char const* argv[])
{
    /*
        1. int m - A matrix rows & C matrix rows
        2. int n - B matrix columns & C matrix columns
        3. int k - A matrix columns & B matrix rows
        4. float alpha
        5. float beta
        6. const char* - A matrix file
        7. const char* - B matrix file
        8. const char* - C matrix file
        9. const char* - output matrix file
        
        example:
        ./e 3 3 3 1 1 A.txt B.txt C.txt out.txt

        A: m*k
        B: k*n
        C: m*n
    */
    int m = atoi(argv[1]);
    int n = atoi(argv[2]);
    int k = atoi(argv[3]);

    float alpha = atof(argv[4]);
    float beta = atof(argv[5]);

    std::ifstream f0(argv[6]);
    std::ifstream f1(argv[7]);
    std::ifstream f2(argv[8]);
    std::ofstream f3(argv[9]);

    hipblasHandle_t handle;

    float* a;
    float* b;
    float* c;

    a = (float*)malloc(m * k * sizeof(float));
    b = (float*)malloc(k * n * sizeof(float));
    c = (float*)malloc(m * n * sizeof(float));

    // read files
    float t;
    int i, j;
    for (i = 0; i < m; i++) {
        for (j = 0; j < k; j++) {
            f0 >> t;
            a[i * k + j] = t;
        }
    }
    for (i = 0; i < k; i++) {
        for (j = 0; j < n; j++) {
            f1 >> t;
            b[i * n + j] = t;
        }
    }
    for (i = 0; i < m; i++) {
        for (j = 0; j < n; j++) {
            f2 >> t;
            c[i * n + j] = t;
        }
    }

    float* d_a;
    float* d_b;
    float* d_c;

    // alocate memory on device
    HANDLE_ERROR(hipMalloc((void**)&d_a, m * k * sizeof(*a)));
    HANDLE_ERROR(hipMalloc((void**)&d_b, k * n * sizeof(*b)));
    HANDLE_ERROR(hipMalloc((void**)&d_c, m * n * sizeof(*c)));

    // create context
    HANDLE_BLERR(hipblasCreate(&handle));
    
    // copy host to device
    HANDLE_BLERR(hipblasSetMatrix(m, k, sizeof(*a), a, m, d_a, m));
    HANDLE_BLERR(hipblasSetMatrix(k, n, sizeof(*b), b, k, d_b, k));
    HANDLE_BLERR(hipblasSetMatrix(m, n, sizeof(*c), c, m, d_c, m));

    // matrix multiplication
    HANDLE_BLERR(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_a, m, d_b, k, &beta, d_c, m));

    // copy device to host
    HANDLE_BLERR(hipblasGetMatrix(m, n, sizeof(*c), d_c, m, c, m));

    // write file
    for (i = 0; i < m; i++) {
        for (j = 0; j < n; j++) {
            #ifdef FLOAT_IO_REPR
            f3 << FLOAT_IO_REPR << c[i * n + j] << " ";
            #else
            f3 << c[i * n + j] << " ";
            #endif
        }
        f3 << std::endl;
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipblasDestroy(handle);

    return 0;
}
